#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
using namespace std;

__global__ void AddInts(int *a, int *b, int count){
	// Create a unique index for each thread
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// Check id if within the bounds of count and add only those items
	if(id < count){
		a[id] += b[id];
	}
}


int main(){
	srand(time(NULL));
	int count = 100;
	int *h_a = new int[count];
	int *h_b = new int[count];

	for (int i = 0; i<count; i++){
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000; 
	}

	cout << "Prior to addition: "<< endl;
	for(int i = 0; i < 5; i++){
		cout << h_a[i] << " " << h_b[i] << endl;
	}

	// Device copies of the arrays
	int *d_a, *d_b;

	if(hipMalloc(&d_a, sizeof(int)*count) != hipSuccess){
		cout << "Error in memory allocation of array A.";
		return 0;
	}

	if(hipMalloc(&d_b, sizeof(int)*count) != hipSuccess){
		cout << "Error in memory allocation of array B.";
		hipFree(d_a);
		return 0;
	}

	// Copy array contents from host to device
	if(hipMemcpy(d_a, h_a, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess){
		cout << "Could not copy array A."<<endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	if(hipMemcpy(d_b, h_b, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess){
		cout << "Could not copy array B."<<endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	// Initialize kernel function
	// In this case only 1 block but 256 threads
	AddInts <<< count / 256 + 1, 256 >>> (d_a, d_b,count);


	// Copy answer to host
	if (hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost) != hipSuccess){
		cout << "Could not copy from device!"<<endl;
		delete[] h_a;
		delete[] h_b;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	for(int i=0; i<5; i++){
		cout << "It's "<<h_a[i] << endl;
	}


	hipFree(d_a);
	hipFree(d_b);
	delete[] h_a;
	delete[] h_b;

	return 0;
}