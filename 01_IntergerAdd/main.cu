#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void AddIntsCuda(int *a, int *b){
	a[0] +=b[0]*b[0];
}

// Basic implementation of CUDA

int main(){
	
	// Initialize host variables
	int h_a=2, h_b=3;
	int *d_a, *d_b; //Device pointer

	// Allocate memory to device
	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));


	// Copy to device memory
	hipMemcpy(d_a,&h_a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b,&h_b, sizeof(int), hipMemcpyHostToDevice);

	// Call the kernel
	AddIntsCuda<<<1,1>>>(d_a,d_b);

	// Copy answer from GPU to HOST
	hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	// Print from host
	cout<<"Answer "<<h_a<<endl;

	// Free memory
	hipFree(d_a);
	hipFree(d_b);

	return 0;
}