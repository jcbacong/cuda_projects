#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void AddIntsCuda(int *a, int *b){
for(int i = 0; i<10000005; i++)
	a[0] +=b[0]*b[0];
}


int main(){
	
	int h_a=2, h_b=3;
	int *d_a, *d_b; //Device pointer

	// Allocate memory to device
	if(hipMalloc((void**)&d_a, sizeof(int))!=hipSuccess){
		cout << "Error allocating memory!"<<endl;
		return 0;
	}

	if(hipMalloc(&d_b, sizeof(int)) !=hipSuccess){
		cout << "Error allocating memory!"<<endl;
		hipFree(d_a);
		return 0;
	}


	// Copy to device memory
	if(hipMemcpy(d_a,&h_a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
		cout << "Error copying memory!"<<endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}
		if(hipMemcpy(d_b,&h_b, sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
		cout << "Error copying memory!"<<endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	// Call the kernel
	AddIntsCuda<<<1,1>>>(d_a,d_b);

	// Copy answer from GPU to HOST
	if(hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
		cout << "Error copying to memory!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	// Print from host
	cout<<"Answer "<<h_a<<endl;

	// Free memory
	hipFree(d_a);
	hipFree(d_b);

	hipDeviceReset();

	return 0;
}