#include "hip/hip_runtime.h"
#include <iostream>
#include "closestGPU.h"
#include "closestCPU.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

int main(){

	// Number of points
	const int count = 100000;

	// Create an array of points and floats (HOST)
	int* index = new int[count];
	float3* Newpoints = new float3[count];
	for(int i=0; i<count; i++){
		Newpoints[i].x = (float)((rand()%10000)-5000);
		Newpoints[i].y = (float)((rand()%10000)-5000);
		Newpoints[i].z = (float)((rand()%10000)-5000);
	}

	// Create an array of points and floats (DEVICE)
	int* d_index;
	float3* d_Newpoints;
	
	// Allocate memory to GPU
	if(hipMalloc(&d_index, sizeof(int)*count) != hipSuccess){
		cout << "Error in memory allocation of array index.";
		return 0;
	}

	if(hipMalloc(&d_Newpoints, sizeof(float3)*count) != hipSuccess){
		cout << "Error in memory allocation of array Newpoints.";
		hipFree(d_index);
		return 0;
	}

	// Copy to device memory
	hipMemcpy(d_index,index, sizeof(int)*count, hipMemcpyHostToDevice);
	hipMemcpy(d_Newpoints, Newpoints, sizeof(float3)*count, hipMemcpyHostToDevice);

	// Upper bound of fastest time
	long fastest = 1000000;
	for(int j=0; j<20; j++){

		// Start time
		long start = clock();

		// Run the algorithm
		// findClosestCPU(Newpoints, index, count); // CPU
		findClosestGPU <<<(count/32)+1,32>>> (d_Newpoints, d_index,count);
		hipMemcpy(index, d_index, sizeof(int)*count, hipMemcpyDeviceToHost);

		// End time
		long finish = clock();

		cout <<"Run "<<j<<" took "<<(finish-start)<<" millis "<<endl;

		if((finish-start) < fastest){
			fastest = (finish-start);
		}
	}

	

	// Print fastest time
	cout<<"======================="<<endl;
	cout << "Fastest time: "<<fastest<<" millis "<<endl;

	// Print results
	cout<<"+++++++++++++++++++++++"<<endl;
	cout<< "Printing 5 sample results..."<<endl;
	for(int i=0; i<5; i++){
		cout<<i<<" --> ("<< Newpoints[i].x<< ","<<Newpoints[i].y<<","<<Newpoints[i].z<<")"<<endl;
	}

	cout<<"+++++++++++++++++++++++"<<endl;
	cout<< "Printing 5 sample results..."<<endl;
	for(int i=0; i<5; i++){
		cout<<i<<" --> "<< index[i]<< endl;
	}


	// Deallocate ram
	hipFree(d_Newpoints);
	hipFree(d_index);
	delete[] index;
	delete[] Newpoints;


	return 0;
}